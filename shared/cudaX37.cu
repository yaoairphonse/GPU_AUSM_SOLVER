#include "hip/hip_runtime.h"
#include"gpu.h"




__global__ void neighbor(float *U,float *UR,float *UL,float *UT,float *UD,float *UF,float *UB,float *body);
__global__ void Unew(float *U,float *UR,float *UL,float *UT,float *UD,float *UF,float *UB);

__device__ float cor_M(float Vin,float Vout,float Tin,float Tout){
float Min,Mout;
float ain,aout;
float M_plus,M_sub;

ain=sqrtf(GAMA*R*Tin);
aout=sqrtf(GAMA*R*Tout);

Min=Vin/ain;
Mout=Vout/aout;

        if(fabs(Min)>1){
                M_plus=0.5*(Min+fabs(Min));
        }else{
                M_plus=0.25*(Min+1)*(Min+1);
        }
        if(fabs(Mout)>1){
                M_sub=0.5*(Mout-fabs(Mout));
        }else{
                M_sub=-0.25*(Mout-1)*(Mout-1);
        }

                return M_plus+M_sub;
}
__device__ float cor_P(float rhoin,float rhoout,float Vin,float Vout ,float Tin,float Tout){

float Min,Mout;
float ain,aout;
float P_plus,P_sub;
float Pin,Pout;

ain=sqrtf(GAMA*R*Tin);
aout=sqrtf(GAMA*R*Tout);

Min=Vin/ain;
Mout=Vout/aout;

Pin=rhoin*R*Tin;
Pout=rhoout*R*Tout;

        if(fabs(Min)>1){
                P_plus=0.5*((Min+fabs(Min))/Min)*Pin;
        }else{
                P_plus=0.25*(Min+1)*(Min+1)*(2-Min)*Pin;
        }
        if(fabs(Mout)>1){
                P_sub=0.5*((Mout-fabs(Mout))/Mout)*Pout;
        }else{
                P_sub=0.25*(Mout-1)*(Mout-1)*(2+Mout)*Pout;
        }

        return P_plus+P_sub;
}

__device__ float flux(float A,float T){
//A is properties V is velocity T is temperature
float face;
float sonic;

sonic=sqrtf(GAMA*R*T);

face=sonic*A;

return face;

}

__device__ float fluxT(float pE,float T,float rho){
//pE mean rho*E
float face2;
float sonic;

sonic=sqrtf(GAMA*R*T);

face2=sonic*(pE+rho*R*T);

return face2;
}


void Call_Unew(){
int threadsPerBlock =512 ;
int blocksPerGrid =(N+threadsPerBlock-1)/threadsPerBlock ;
//size_t size;
Unew<<<blocksPerGrid, threadsPerBlock>>>(d_U,d_UR,d_UL,d_UT,d_UD,d_UF,d_UB);

hipDeviceSynchronize();
}

__global__ void Unew(float *U,float *UR,float *UL,float *UT,float *UD,float *UF,float *UB){
int i= blockDim.x * blockIdx.x +threadIdx.x ;
//int x_cell,y_cell,z_cell;
//int five=5;
float Mach;
float press;
float Pc[5],PR[5],PL[5],PT[5],PD[5],PF[5],PB[5]; // properties
float FR[5],FL[5],FT[5],FD[5],FF[5],FB[5];
		if(i<N){
				//z_cell =(int)i/(NX*NY);
				//y_cell =(int)(i-z_cell*NY*NX)/NX;
				//x_cell = i-y_cell*NX-z_cell*NX*NY;				
				Pc[0]=U[5*i];   
				PR[0]=UR[5*i];
				PL[0]=UL[5*i]; 
				PT[0]=UT[5*i]; 
				PD[0]=UD[5*i]; 
				PF[0]=UF[5*i]; 
				PB[0]=UB[5*i];
				//ux
				Pc[1]=U[5*i+1]/U[5*i];	
				PR[1]=UR[5*i+1]/UR[5*i];
				PL[1]=UL[5*i+1]/UL[5*i]; 
				PT[1]=UT[5*i+1]/UT[5*i]; 
				PD[1]=UD[5*i+1]/UD[5*i]; 
				PF[1]=UF[5*i+1]/UF[5*i]; 
				PB[1]=UB[5*i+1]/UB[5*i];	
				//uy
				Pc[2]=U[5*i+2]/U[5*i];
				PR[2]=UR[5*i+2]/UR[5*i];
				PL[2]=UL[5*i+2]/UL[5*i]; 
				PT[2]=UT[5*i+2]/UT[5*i]; 
				PD[2]=UD[5*i+2]/UD[5*i]; 
				PF[2]=UF[5*i+2]/UF[5*i];
				PB[2]=UB[5*i+2]/UB[5*i];	
				//uz
				Pc[3]=U[5*i+3]/U[5*i]; 
				PR[3]=UR[5*i+3]/UR[5*i];
				PL[3]=UL[5*i+3]/UL[5*i]; 
				PT[3]=UT[5*i+3]/UT[5*i]; 
				PD[3]=UD[5*i+3]/UD[5*i]; 
				PF[3]=UF[5*i+3]/UF[5*i]; 
				PB[3]=UB[5*i+3]/UB[5*i];
				//temp...
				Pc[4]=((U[5*i+4]/U[5*i])-0.5*(Pc[1]*Pc[1]+Pc[2]*Pc[2]+Pc[3]*Pc[3]))/Cv; 
				PR[4]=((UR[5*i+4]/UR[5*i])-0.5*(PR[1]*PR[1]+PR[2]*PR[2]+PR[3]*PR[3]))/Cv;
				PL[4]=((UL[5*i+4]/UL[5*i])-0.5*(PL[1]*PL[1]+PL[2]*PL[2]+PL[3]*PL[3]))/Cv; 
				PT[4]=((UT[5*i+4]/UT[5*i])-0.5*(PT[1]*PT[1]+PT[2]*PT[2]+PT[3]*PT[3]))/Cv; 
				PD[4]=((UD[5*i+4]/UD[5*i])-0.5*(PD[1]*PD[1]+PD[2]*PD[2]+PD[3]*PD[3]))/Cv; 
				PF[4]=((UF[5*i+4]/UF[5*i])-0.5*(PF[1]*PF[1]+PF[2]*PF[2]+PF[3]*PF[3]))/Cv; 
				PB[4]=((UB[5*i+4]/UB[5*i])-0.5*(PB[1]*PB[1]+PB[2]*PB[2]+PB[3]*PB[3]))/Cv;
				
				
				//right
		Mach=cor_M(Pc[1],PR[1],Pc[4],PR[4]); press=cor_P(Pc[0],PR[0],Pc[1],PR[1],Pc[4],PR[4]);
		FR[0]=0.5*Mach*(flux(UR[5*i],PR[4])+flux(U[5*i],Pc[4]))-0.5*fabs(Mach)*(flux(UR[5*i],PR[4])-flux(U[5*i],Pc[4]));
		FR[1]=0.5*Mach*(flux(UR[5*i+1],PR[4])+flux(U[5*i+1],Pc[4]))-0.5*fabs(Mach)*(flux(UR[5*i+1],PR[4])-flux(U[5*i+1],Pc[4]))+press;
		FR[2]=0.5*Mach*(flux(UR[5*i+2],PR[4])+flux(U[5*i+2],Pc[4]))-0.5*fabs(Mach)*(flux(UR[5*i+2],PR[4])-flux(U[5*i+2],Pc[4]));
		FR[3]=0.5*Mach*(flux(UR[5*i+3],PR[4])+flux(U[5*i+3],Pc[4]))-0.5*fabs(Mach)*(flux(UR[5*i+3],PR[4])-flux(U[5*i+3],Pc[4]));
		FR[4]=0.5*Mach*(fluxT(UR[5*i+4],PR[4],PR[0])+fluxT(U[5*i+4],Pc[4],Pc[0]))-0.5*fabs(Mach)*(fluxT(UR[5*i+4],PR[4],PR[0])-fluxT(U[5*i+4],Pc[4],Pc[0]));
				//left
                Mach=cor_M(PL[1],Pc[1],PL[4],Pc[4]); press=cor_P(PL[0],Pc[0],PL[1],Pc[1],PL[4],Pc[4]);
                FL[0]=0.5*Mach*(flux(UL[5*i],PL[4])+flux(U[5*i],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i],Pc[4])-flux(UL[5*i],PL[4]));
                FL[1]=0.5*Mach*(flux(UL[5*i+1],PL[4])+flux(U[5*i+1],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i+1],Pc[4])-flux(UL[5*i+1],PL[4]))+press;
                FL[2]=0.5*Mach*(flux(UL[5*i+2],PL[4])+flux(U[5*i+2],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i+2],Pc[4])-flux(UL[5*i+2],PL[4]));
                FL[3]=0.5*Mach*(flux(UL[5*i+3],PL[4])+flux(U[5*i+3],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i+3],Pc[4])-flux(UL[5*i+3],PL[4]));
                FL[4]=0.5*Mach*(fluxT(UL[5*i+4],PL[4],PL[0])+fluxT(U[5*i+4],Pc[4],Pc[0]))-0.5*fabs(Mach)*(fluxT(U[5*i+4],Pc[4],Pc[0])-fluxT(UL[5*i+4],PL[4],PL[0]));
				//top
                Mach=cor_M(Pc[2],PT[2],Pc[4],PT[4]); press=cor_P(Pc[0],PT[0],Pc[2],PT[2],Pc[4],PT[4]);
                FT[0]=0.5*Mach*(flux(UT[5*i],PT[4])+flux(U[5*i],Pc[4]))-0.5*fabs(Mach)*(flux(UT[5*i],PT[4])-flux(U[5*i],Pc[4]));
                FT[1]=0.5*Mach*(flux(UT[5*i+1],PT[4])+flux(U[5*i+1],Pc[4]))-0.5*fabs(Mach)*(flux(UT[5*i+1],PT[4])-flux(U[5*i+1],Pc[4]));
                FT[2]=0.5*Mach*(flux(UT[5*i+2],PT[4])+flux(U[5*i+2],Pc[4]))-0.5*fabs(Mach)*(flux(UT[5*i+2],PT[4])-flux(U[5*i+2],Pc[4]))+press;
                FT[3]=0.5*Mach*(flux(UT[5*i+3],PT[4])+flux(U[5*i+3],Pc[4]))-0.5*fabs(Mach)*(flux(UT[5*i+3],PT[4])-flux(U[5*i+3],Pc[4]));
                FT[4]=0.5*Mach*(fluxT(UT[5*i+4],PT[4],PT[0])+fluxT(U[5*i+4],Pc[4],Pc[0]))-0.5*fabs(Mach)*(fluxT(UT[5*i+4],PT[4],PT[0])-fluxT(U[5*i+4],Pc[4],Pc[0]));
				//DOWN
                Mach=cor_M(PD[2],Pc[2],PD[4],Pc[4]); press=cor_P(PD[0],Pc[0],PD[2],Pc[2],PD[4],Pc[4]);
                FD[0]=0.5*Mach*(flux(UD[5*i],PD[4])+flux(U[5*i],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i],Pc[4])-flux(UD[5*i],PD[4]));
                FD[1]=0.5*Mach*(flux(UD[5*i+1],PD[4])+flux(U[5*i+1],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i+1],Pc[4])-flux(UD[5*i+1],PD[4]));
                FD[2]=0.5*Mach*(flux(UD[5*i+2],PD[4])+flux(U[5*i+2],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i+2],Pc[4])-flux(UD[5*i+2],PD[4]))+press;
                FD[3]=0.5*Mach*(flux(UD[5*i+3],PD[4])+flux(U[5*i+3],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i+3],Pc[4])-flux(UD[5*i+3],PD[4]));
                FD[4]=0.5*Mach*(fluxT(UD[5*i+4],PD[4],PD[0])+fluxT(U[5*i+4],Pc[4],Pc[0]))-0.5*fabs(Mach)*(fluxT(U[5*i+4],Pc[4],Pc[0])-fluxT(UD[5*i+4],PD[4],PD[0]));
				//front
                Mach=cor_M(Pc[3],PF[3],Pc[4],PF[4]); press=cor_P(Pc[0],PF[0],Pc[3],PF[3],Pc[4],PF[4]);
                FF[0]=0.5*Mach*(flux(UF[5*i],PF[4])+flux(U[5*i],Pc[4]))-0.5*fabs(Mach)*(flux(UF[5*i],PF[4])-flux(U[5*i],Pc[4]));
                FF[1]=0.5*Mach*(flux(UF[5*i+1],PF[4])+flux(U[5*i+1],Pc[4]))-0.5*fabs(Mach)*(flux(UF[5*i+1],PF[4])-flux(U[5*i+1],Pc[4]));
                FF[2]=0.5*Mach*(flux(UF[5*i+2],PF[4])+flux(U[5*i+2],Pc[4]))-0.5*fabs(Mach)*(flux(UF[5*i+2],PF[4])-flux(U[5*i+2],Pc[4]));
                FF[3]=0.5*Mach*(flux(UF[5*i+3],PF[4])+flux(U[5*i+3],Pc[4]))-0.5*fabs(Mach)*(flux(UF[5*i+3],PF[4])-flux(U[5*i+3],Pc[4]))+press;
                FF[4]=0.5*Mach*(fluxT(UF[5*i+4],PF[4],PF[0])+fluxT(U[5*i+4],Pc[4],Pc[0]))-0.5*fabs(Mach)*(fluxT(UF[5*i+4],PF[4],PF[0])-fluxT(U[5*i+4],Pc[4],Pc[0]));
				//back	
                Mach=cor_M(PB[3],Pc[3],PB[4],Pc[4]); press=cor_P(PB[0],Pc[0],PB[3],Pc[3],PB[4],Pc[4]);
                FB[0]=0.5*Mach*(flux(UB[5*i],PB[4])+flux(U[5*i],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i],Pc[4])-flux(UB[5*i],PB[4]));
                FB[1]=0.5*Mach*(flux(UB[5*i+1],PB[4])+flux(U[5*i+1],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i+1],Pc[4])-flux(UB[5*i+1],PB[4]));
                FB[2]=0.5*Mach*(flux(UB[5*i+2],PB[4])+flux(U[5*i+2],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i+2],Pc[4])-flux(UB[5*i+2],PB[4]));
                FB[3]=0.5*Mach*(flux(UB[5*i+3],PB[4])+flux(U[5*i+3],Pc[4]))-0.5*fabs(Mach)*(flux(U[5*i+3],Pc[4])-flux(UB[5*i+3],PB[4]))+press;
                FB[4]=0.5*Mach*(fluxT(UB[5*i+4],PB[4],PB[0])+fluxT(U[5*i+4],Pc[4],Pc[0]))-0.5*fabs(Mach)*(fluxT(U[5*i+4],Pc[4],Pc[0])-fluxT(UB[5*i+4],PB[4],PB[0]));
		

				//examine that one time step renew a range with five  (5*i)+0~(5*i+4),no neighbor to interfere.
				//renew
				 U[5*i]=U[5*i]-(DT/DX)*(FR[0]-FL[0])-(DT/DY)*(FT[0]-FD[0])-(DT/DZ)*(FF[0]-FB[0]);
				 U[5*i+1]=U[5*i+1]-(DT/DX)*(FR[1]-FL[1])-(DT/DY)*(FT[1]-FD[1])-(DT/DZ)*(FF[1]-FB[1]);
				 U[5*i+2]=U[5*i+2]-(DT/DX)*(FR[2]-FL[2])-(DT/DY)*(FT[2]-FD[2])-(DT/DZ)*(FF[2]-FB[2]);
				 U[5*i+3]=U[5*i+3]-(DT/DX)*(FR[3]-FL[3])-(DT/DY)*(FT[3]-FD[3])-(DT/DZ)*(FF[3]-FB[3]);
				 U[5*i+4]=U[5*i+4]-(DT/DX)*(FR[4]-FL[4])-(DT/DY)*(FT[4]-FD[4])-(DT/DZ)*(FF[4]-FB[4]);

			        

		
		}
}

void Call_neighbor(){
int threadsPerBlock =512 ;
int blocksPerGrid =(N+threadsPerBlock-1)/threadsPerBlock ;
//size_t size;
neighbor<<<blocksPerGrid, threadsPerBlock>>>(d_U,d_UR,d_UL,d_UT,d_UD,d_UF,d_UB,d_body);

hipDeviceSynchronize();


}

__global__ void neighbor(float *U,float *UR,float *UL,float *UT,float *UD,float *UF,float *UB,float *body){

int i= blockDim.x * blockIdx.x +threadIdx.x ;
int cx,cy,cz;
int five=5;
	if(i<N){
	        cz =(int)i/(NX*NY);
         	cy =(int)(i-cz*NY*NX)/NX;
         	cx = i-cy*NX-cz*NX*NY;	
			if(cx<(NX-1)){
					if(body[i+1]<0.1){
					//air
						UR[5*i]=U[5*i+five];
						UR[5*i+1]=U[5*i+1+five];
						UR[5*i+2]=U[5*i+2+five];
						UR[5*i+3]=U[5*i+3+five];
						UR[5*i+4]=U[5*i+4+five];
							}else{
							//body
										UR[5*i]=U[5*i];
										UR[5*i+1]=-U[5*i+1];
										UR[5*i+2]=U[5*i+2];
										UR[5*i+3]=U[5*i+3];
										UR[5*i+4]=U[5*i+4];
						}
			}else{
			//right boundary,outlet~
			UR[5*i]=U[5*i];
			UR[5*i+1]=U[5*i+1];
			UR[5*i+2]=U[5*i+2];
			UR[5*i+3]=U[5*i+3];
			UR[5*i+4]=U[5*i+4];
			}
			
			
		 	if(cx>0){
				if(body[i-1]<0.1){
					UL[5*i]=U[5*i-five];
					UL[5*i+1]=U[5*i+1-five];
					UL[5*i+2]=U[5*i+2-five];
					UL[5*i+3]=U[5*i+3-five];
					UL[5*i+4]=U[5*i+4-five];
					}else{
								UL[5*i]=U[5*i];
								UL[5*i+1]=-U[5*i+1];
								UL[5*i+2]=U[5*i+2];
								UL[5*i+3]=U[5*i+3];
								UL[5*i+4]=U[5*i+4];
						}
			}else{
			UL[5*i]=U[5*i];
			UL[5*i+1]=U[5*i+1];
			UL[5*i+2]=U[5*i+2];
			UL[5*i+3]=U[5*i+3];
			UL[5*i+4]=U[5*i+4];				
			}
			
		
			if(cy<NY-1){
			
				if(body[i+NX]<0.1){
				    UT[5*i]=U[5*i+five*NX];
					UT[5*i+1]=U[5*i+1+five*NX];
					UT[5*i+2]=U[5*i+2+five*NX];
					UT[5*i+3]=U[5*i+3+five*NX];
					UT[5*i+4]=U[5*i+4+five*NX];
					}else{
							UT[5*i]=U[5*i];
							UT[5*i+1]=U[5*i+1];
							UT[5*i+2]=-U[5*i+2];
							UT[5*i+3]=U[5*i+3];
							UT[5*i+4]=U[5*i+4];
						}	
			}else{
					UT[5*i]=U[5*i];
					UT[5*i+1]=U[5*i+1];
					UT[5*i+2]=U[5*i+2];
					UT[5*i+3]=U[5*i+3];
					UT[5*i+4]=U[5*i+4];			
			
			
			
			}
		 	if(cy>0){
				if(body[i-NX]<0.1){
					UD[5*i]=U[5*i-five*NX];
					UD[5*i+1]=U[5*i+1-five*NX];
					UD[5*i+2]=U[5*i+2-five*NX];
					UD[5*i+3]=U[5*i+3-five*NX];
					UD[5*i+4]=U[5*i+4-five*NX];
					}else{
					UD[5*i]=U[5*i];
					UD[5*i+1]=U[5*i+1];
					UD[5*i+2]=-U[5*i+2];
					UD[5*i+3]=U[5*i+3];
					UD[5*i+4]=U[5*i+4];
						}
			}else{
			//inlet T=213k rho=1e-4
                                        UD[5*i]=U[5*i];
                                        UD[5*i+1]=U[5*i+1];
                                        UD[5*i+2]=U[5*i+2];
                                        UD[5*i+3]=U[5*i+3];
                                        UD[5*i+4]=U[5*i+4];

			}
			if(cz<NZ-1){
				if(body[i+NX*NY]<0.1){
					UF[5*i]=U[5*i+five*NX*NY];
					UF[5*i+1]=U[5*i+1+five*NX*NY];
					UF[5*i+2]=U[5*i+2+five*NX*NY];
					UF[5*i+3]=U[5*i+3+five*NX*NY];
					UF[5*i+4]=U[5*i+4+five*NX*NY];
					}else{
					UF[5*i]=U[5*i];
					UF[5*i+1]=U[5*i+1];
					UF[5*i+2]=U[5*i+2];
					UF[5*i+3]=-U[5*i+3];
					UF[5*i+4]=U[5*i+4];
					}
			}else{
					UF[5*i]=U[5*i];
					UF[5*i+1]=U[5*i+1];
					UF[5*i+2]=U[5*i+2];
					UF[5*i+3]=U[5*i+3];
					UF[5*i+4]=U[5*i+4];
			
			
			}
			if(cz>0){
			
				if(body[i-NX*NY]<0.1){
					UB[5*i]=U[5*i-five*NX*NY];
					UB[5*i+1]=U[5*i+1-five*NX*NY];
					UB[5*i+2]=U[5*i+2-five*NX*NY];
					UB[5*i+3]=U[5*i+3-five*NX*NY];
					UB[5*i+4]=U[5*i+4-five*NX*NY];
					}else{
					UB[5*i]=U[5*i];
					UB[5*i+1]=U[5*i+1];
					UB[5*i+2]=U[5*i+2];
					UB[5*i+3]=-U[5*i+3];
					UB[5*i+4]=U[5*i+4];
						}	
			}else{
					UB[5*i]=U[5*i];
					UB[5*i+1]=U[5*i+1];
					UB[5*i+2]=U[5*i+2];
					UB[5*i+3]=U[5*i+3];
					UB[5*i+4]=U[5*i+4];		
					
			}
	}

}





void Send_To_Device() {
// Size of data to send

size_t size ;
hipError_t Error ;

	 size = N*sizeof(float);
         Error = hipMemcpy(d_body,h_body,size,hipMemcpyHostToDevice);
         printf("CUDA error(memcpy h_body -> d_body)=%s\n",hipGetErrorString(Error));


	 size = 5*N*sizeof(float);
//	 Error = hipMemcpy(d_P,h_P,size,hipMemcpyHostToDevice);
//	 printf("CUDA error(memcpy h_P -> d_P)=%s\n",hipGetErrorString(Error));
	 Error = hipMemcpy(d_U,h_U,size,hipMemcpyHostToDevice);
         printf("CUDA error(memcpy h_U -> d_U)=%s\n",hipGetErrorString(Error));

}
void Get_From_Device(){


size_t size ;
hipError_t Error ;

     size = N*sizeof(float);
  Error= hipMemcpy(h_body,d_body,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_body->h_body)=%s\n",hipGetErrorString(Error));



  size=5*N*sizeof(float);
  
  Error= hipMemcpy(h_U,d_U,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_U->h_U)=%s\n",hipGetErrorString(Error));
//  Error= hipMemcpy(h_P,d_P,size,hipMemcpyDeviceToHost);
//  printf("CUDA error(memcpy d_P->h_P)=%s\n",hipGetErrorString(Error));

    Error= hipMemcpy(h_UR,d_UR,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_U->h_U)=%s\n",hipGetErrorString(Error));
  Error= hipMemcpy(h_UL,d_UL,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_U->h_U)=%s\n",hipGetErrorString(Error));
  Error= hipMemcpy(h_UT,d_UT,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_U->h_U)=%s\n",hipGetErrorString(Error));
  Error= hipMemcpy(h_UD,d_UD,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_U->h_U)=%s\n",hipGetErrorString(Error));
  Error= hipMemcpy(h_UF,d_UF,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_U->h_U)=%s\n",hipGetErrorString(Error));
  Error= hipMemcpy(h_UB,d_UB,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_U->h_U)=%s\n",hipGetErrorString(Error));


}

void Allocate_Memory(){
 
   size_t size ;
   hipError_t Error ;

   size=N*sizeof(float) ;
     h_body = (float*)malloc(size) ;
    Error =hipMalloc((void**)&d_body,size);
   printf("CUDA error (malloc d_body)=%s\n",hipGetErrorString(Error));

   size= 5*N*sizeof(float) ;
   
   

   h_P = (float*)malloc(size) ;  
   h_U = (float*)malloc(size) ;
   h_UR = (float*)malloc(size) ;
   h_UL = (float*)malloc(size) ;
   h_UT = (float*)malloc(size) ;
   h_UD = (float*)malloc(size) ;
   h_UF = (float*)malloc(size) ;
   h_UB = (float*)malloc(size) ;



  // Error =hipMalloc((void**)&d_P,size);
  // printf("CUDA error (malloc d_P)=%s\n",hipGetErrorString(Error));
   Error =hipMalloc((void**)&d_U,size);
   printf("CUDA error (malloc d_U)=%s\n",hipGetErrorString(Error));
   Error =hipMalloc((void**)&d_UR,size);
   printf("CUDA error (malloc d_UR)=%s\n",hipGetErrorString(Error));
   Error =hipMalloc((void**)&d_UL,size);
   printf("CUDA error (malloc d_UL)=%s\n",hipGetErrorString(Error));
   Error =hipMalloc((void**)&d_UT,size);
   printf("CUDA error (malloc d_UT)=%s\n",hipGetErrorString(Error));
   Error =hipMalloc((void**)&d_UD,size);
   printf("CUDA error (malloc d_UD)=%s\n",hipGetErrorString(Error));
   Error =hipMalloc((void**)&d_UF,size);
   printf("CUDA error (malloc d_UF)=%s\n",hipGetErrorString(Error));
   Error =hipMalloc((void**)&d_UB,size);
   printf("CUDA error (malloc d_UB)=%s\n",hipGetErrorString(Error));


}

void Free_Memory(){
	if (h_body) free(h_body);
	if (d_body) hipFree(d_body);
        if (h_U) free(h_U);
        if (h_P) free(h_P);
      //  if (d_P) hipFree(d_P);
	if (d_U) hipFree(d_U);
	if (d_UR) hipFree(d_UR);
 	if (d_UL) hipFree(d_UL);
 	if (d_UT) hipFree(d_UT);
 	if (d_UD) hipFree(d_UD);
 	if (d_UF) hipFree(d_UF);
 	if (d_UB) hipFree(d_UB);

        if (h_UR) free(h_UR);
        if (h_UL) free(h_UL);
        if (h_UT) free(h_UT);
        if (h_UD) free(h_UD);
        if (h_UF) free(h_UF);
        if (h_UB) free(h_UB);



}
