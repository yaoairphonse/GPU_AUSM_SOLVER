#include "hip/hip_runtime.h"
#include"gpu.h"

__global__ void neighbor(float *U,float *UR,float *UL);
__global__ void Unew(float *U,float *UR,float *UL);





__device__ float Rusanov(float U_right,float U_left ,float F_right ,float F_left,float speed){

  
return 0.5*(F_right+F_left)-speed*(U_right-U_left);

}

void Call_Unew(){
int threadsPerBlock =512 ;
int blocksPerGrid =(N+threadsPerBlock-1)/threadsPerBlock ;
//size_t size;
Unew<<<blocksPerGrid, threadsPerBlock>>>(d_U,d_UR,d_UL);

}



__global__ void Unew(float *U,float *UR,float *UL){
int I= threadIdx.x;
int i= blockDim.x * blockIdx.x +I ;
float right,left;
float wave;
float FR[2],FL[2];
float Velc,VelR,VelL;
__shared__ float UR1_s[512],UR2_s[512],UL1_s[512],UL2_s[512],U1_s[512],U2_s[512];

	UR1_s[I]=UR[2*i];
        UR2_s[I]=UR[2*i+1];
        U1_s[I]=U[2*i];
        U2_s[I]=U[2*i+1];
        UL1_s[I]=UL[2*i];
        UL2_s[I]=UL[2*i+1];


	if(I<128){
		
		VelR=UR2_s[I]/UR1_s[I];
		Velc=U2_s[I]/U1_s[I];
		VelL=UL2_s[I]/UL1_s[I];
		
	//right
		wave=sqrtf((UR1_s[I]+U1_s[I])*0.5*g);
		right=UR1_s[I]*VelR   ;left=U1_s[I]*Velc     ;
		FR[0]=Rusanov(UR1_s[I],U1_s[I],right,left,wave);
		right=UR1_s[I]*(VelR*VelR+0.5*g*UR1_s[I])   ;left=U1_s[I]*(Velc*Velc+0.5*g*U1_s[I])     ;
		FR[1]=Rusanov(UR2_s[I],U2_s[I],right,left,wave);
	//left
		wave=sqrtf((U1_s[I]+UL1_s[I])*0.5*g);
		right=U1_s[I]*Velc   ;left=UL1_s[I]*VelL     ;
		FL[0]=Rusanov(U1_s[I],UL1_s[I],right,left,wave);
		right=U1_s[I]*(Velc*Velc+0.5*g*U1_s[I])   ;left=UL1_s[I]*(VelL*VelL+0.5*g*UL1_s[I])     ;
		FL[1]=Rusanov(U2_s[I],UL2_s[I],right,left,wave);

	
		U1_s[I]=U1_s[I]-(DT/DX)*(FR[0]-FL[0]);
		U2_s[I]=U2_s[I]-(DT/DX)*(FR[1]-FL[1]);


	}

	U[2*i]=U1_s[I];
	U[2*i+1]=U2_s[I];
}

void Call_neighbor(){
int threadsPerBlock =128 ;
int blocksPerGrid =(N+threadsPerBlock-1)/threadsPerBlock ;
//size_t size;
neighbor<<<blocksPerGrid, threadsPerBlock>>>(d_U,d_UR,d_UL);

}
__global__ void neighbor(float *U,float *UR,float *UL){

int i= blockDim.x * blockIdx.x +threadIdx.x ;
//__shared__ float UR1_s[128],UR2_s[128],UL1_s[128],UL2_s[128],U1_s[128],U2_s[128];


       // UR1_s[I]=UR[2*i];
       // UR2_s[I]=UR[2*i+1];
        //U1_s[I]=U[2*i];
      //  U2_s[I]=U[2*i+1];
    //    UL1_s[I]=UL[2*i];
  //      UL2_s[I]=UL[2*i+1];
  //      __syncthreads();	

	if(i<N){
	

		if(i<(N-1)){
			UR[2*i]=U[2*(i+1)];
			UR[2*i+1]=U[2*(i+1)+1];
	        
		}else{
			UR[2*i]=U[2*i];
			UR[2*i+1]=-U[2*i+1];
		}
		if(i>0){
			UL[2*i]=U[2*(i-1)];
			UL[2*i+1]=U[2*(i-1)+1];
		}else{
			UL[2*i]=U[2*i];
                        UL[2*i+1]=-U[2*i+1];
	
		}

	}

}


void Free_Memory(){

if (h_U) free(h_U);
if (d_U) hipFree(d_U);
if (d_U) hipFree(d_UR);
if (d_U) hipFree(d_UL);
}

void Allocate_Memory(){
size_t size ;
hipError_t Error ;
size=2*N*sizeof(float);
   h_P=(float*)malloc(size);
   h_U= (float*)malloc(size);

   Error =hipMalloc((void**)&d_U,size);
   printf("CUDA error (malloc d_U)=%s\n",hipGetErrorString(Error));
    Error =hipMalloc((void**)&d_UR,size);
   printf("CUDA error (malloc d_UR)=%s\n",hipGetErrorString(Error));
 Error =hipMalloc((void**)&d_UL,size);
   printf("CUDA error (malloc d_UL)=%s\n",hipGetErrorString(Error));


}

void Send_To_Device(){
size_t size ;
hipError_t Error ;

size=2*N*sizeof(float);


 Error = hipMemcpy(d_U,h_U,size,hipMemcpyHostToDevice);
         printf("CUDA error(memcpy h_U -> d_U)=%s\n",hipGetErrorString(Error));

}

void Get_From_Device(){

size_t size ;
hipError_t Error ;


  size=2*N*sizeof(float);


  Error= hipMemcpy(h_U,d_U,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_U->h_U)=%s\n",hipGetErrorString(Error));


}
