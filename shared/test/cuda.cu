#include "hip/hip_runtime.h"
#include"gpu.h"

__global__ void neighbor(float *U,float *UR,float *UL);
__global__ void Unew(float *U,float *UR,float *UL);





__device__ float Rusanov(float U_right,float U_left ,float F_right ,float F_left,float speed){

  
return 0.5*(F_right+F_left)-speed*(U_right-U_left);

}

void Call_Unew(){
int threadsPerBlock =512 ;
int blocksPerGrid =(N+threadsPerBlock-1)/threadsPerBlock ;
//size_t size;
Unew<<<blocksPerGrid, threadsPerBlock>>>(d_U,d_UR,d_UL);

}



__global__ void Unew(float *U,float *UR,float *UL){
int i= blockDim.x * blockIdx.x +threadIdx.x ;
float right,left;
float wave;
float FR[2],FL[2];
float Velc,VelR,VelL;
	if(i<N){
		
		VelR=UR[2*i+1]/UR[2*i];
		Velc=U[2*i+1]/U[2*i];
		VelL=UL[2*i+1]/UL[2*i];
		
	//right
		wave=sqrtf((UR[2*i]+U[2*i])*0.5*g);
		right=UR[2*i]*VelR   ;left=U[2*i]*Velc     ;
		FR[0]=Rusanov(UR[2*i],U[2*i],right,left,wave);
		right=UR[2*i]*(VelR*VelR+0.5*g*UR[2*i])   ;left=U[2*i]*(Velc*Velc+0.5*g*U[2*i])     ;
		FR[1]=Rusanov(UR[2*i+1],U[2*i+1],right,left,wave);
	//left
		wave=sqrtf((U[2*i]+UL[2*i])*0.5*g);
		right=U[2*i]*Velc   ;left=UL[2*i]*VelL     ;
		FL[0]=Rusanov(U[2*i],UL[2*i],right,left,wave);
		right=U[2*i]*(Velc*Velc+0.5*g*U[2*i])   ;left=UL[2*i]*(VelL*VelL+0.5*g*UL[2*i])     ;
		FL[1]=Rusanov(U[2*i+1],UL[2*i+1],right,left,wave);

	
		U[2*i]=U[2*i]-(DT/DX)*(FR[0]-FL[0]);
		U[2*i+1]=U[2*i+1]-(DT/DX)*(FR[1]-FL[1]);


	}
}
void Call_neighbor(){
int threadsPerBlock =128 ;
int blocksPerGrid =(N+threadsPerBlock-1)/threadsPerBlock ;
//size_t size;
neighbor<<<blocksPerGrid, threadsPerBlock>>>(d_U,d_UR,d_UL);

}
__global__ void neighbor(float *U,float *UR,float *UL){

int i= blockDim.x * blockIdx.x +threadIdx.x ;

	if(i<N){
	

		if(i<(N-1)){
			UR[2*i]=U[2*(i+1)];
			UR[2*i+1]=U[2*(i+1)+1];
	        
		}else{
			UR[2*i]=U[2*i];
			UR[2*i+1]=-U[2*i+1];
		}
		if(i>0){
			UL[2*i]=U[2*(i-1)];
			UL[2*i+1]=U[2*(i-1)+1];
		}else{
			UL[2*i]=U[2*i];
                        UL[2*i+1]=-U[2*i+1];
	
		}

	}

}


void Free_Memory(){

if (h_U) free(h_U);
if (d_U) hipFree(d_U);
if (d_U) hipFree(d_UR);
if (d_U) hipFree(d_UL);
}

void Allocate_Memory(){
size_t size ;
hipError_t Error ;
size=2*N*sizeof(float);
   h_P=(float*)malloc(size);
   h_U= (float*)malloc(size);

   Error =hipMalloc((void**)&d_U,size);
   printf("CUDA error (malloc d_U)=%s\n",hipGetErrorString(Error));
    Error =hipMalloc((void**)&d_UR,size);
   printf("CUDA error (malloc d_UR)=%s\n",hipGetErrorString(Error));
 Error =hipMalloc((void**)&d_UL,size);
   printf("CUDA error (malloc d_UL)=%s\n",hipGetErrorString(Error));


}

void Send_To_Device(){
size_t size ;
hipError_t Error ;

size=2*N*sizeof(float);


 Error = hipMemcpy(d_U,h_U,size,hipMemcpyHostToDevice);
         printf("CUDA error(memcpy h_U -> d_U)=%s\n",hipGetErrorString(Error));

}

void Get_From_Device(){

size_t size ;
hipError_t Error ;


  size=2*N*sizeof(float);


  Error= hipMemcpy(h_U,d_U,size,hipMemcpyDeviceToHost);
  printf("CUDA error(memcpy d_U->h_U)=%s\n",hipGetErrorString(Error));


}
